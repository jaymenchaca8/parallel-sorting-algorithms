#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>


/**
 * mergesort.cu
 * a one-file c++ / cuda program for performing mergesort on the GPU
 * While the program execution is fairly slow, most of its runnning time
 *  is spent allocating memory on the GPU.
 * For a more complex program that performs many calculations,
 *  running on the GPU may provide a significant boost in performance
 */

#define BLOCKS 2
#define THREADS 8
#define ARRAY_SIZE 32
#define min(a, b) (a < b ? a : b)

// data[], size, threads, blocks, 
float mergesort(long*, long, dim3, dim3);
// A[]. B[], size, width, slices, nThreads
__global__ void gpu_mergesort(long*, long*, long, long, long, dim3*, dim3*);
__device__ void gpu_bottomUpMerge(long*, long*, long, long, long);


int main(int argc, char** argv) {
    
    int size = ARRAY_SIZE;
    int arrayType = 0;
    int blocks = BLOCKS;
    int threads = THREADS;
    float tot = 0;
    
    if(argc == 6){                  //./cuda_merge file block threads arraysize arraytype
        blocks = atoi(argv[2]);
        threads = atoi(argv[3]);
        size = atoi(argv[4]);
        arrayType = atoi(argv[5]);
    }
    else{
        printf("args not right");
        exit(-1);
    }
    
    
    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = threads;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    blocksPerGrid.x = blocks;
    blocksPerGrid.y = 1;
    blocksPerGrid.z = 1;

    //
    // Get Unsorted Array
    //

    long* data = (long*)malloc(size * sizeof(long));
    
    FILE *file = fopen(argv[1],"r");
    
    fscanf(file, "%d", &size);
    
    for(int i = 0; i < size; i++){
        fscanf(file, "%d", &data[i]);
    }
    
    fclose(file);
    file = NULL;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    /*
    srand(time(NULL));
    for(int i = 0; i < size; i++){
        data[i] = rand() % size;
        printf("%d ", data[i]);
    }
    printf("\n");
    */
    
    hipEventRecord(start);
    
    // merge-sort the data
    float comm_time = mergesort(data, size, threadsPerBlock, blocksPerGrid);

    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&tot, start, stop);
    
    printf("%d, ", blocks * threads);
    //algorithm - merge cuda -> 3
    printf("%d, ", 3);
    //array type
    printf("%d, ", arrayType);
    //array size
    printf("%d, ", size);
    //total time
    printf("%f, ", tot / 1000); //in seconds
    //communication time
    printf("%f ", comm_time / 1000);
    printf("\n");
    //
    // Print out the list
    //
    /*
    for (int i = 0; i < size; i++) {
        printf("%d ", data[i]);
    } 
    printf("\n");
    */
}

float mergesort(long* data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {

    //
    // Allocate two arrays on the GPU
    // we switch back and forth between them during the sort
    //
    long* D_data;
    long* D_swp;
    dim3* D_threads;
    dim3* D_blocks;
    
    float send, recv;
    
    hipEvent_t start_m, stop_m;
    hipEventCreate(&start_m);
    hipEventCreate(&stop_m);
    
    //send start
    hipEventRecord(start_m);
    // Actually allocate the two arrays
    hipMalloc((void**) &D_data, size * sizeof(long));
    hipMalloc((void**) &D_swp, size * sizeof(long));


    // Copy from our input list into the first array
    hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice);
 
    //
    // Copy the thread / block info to the GPU as well
    //
    hipMalloc((void**) &D_threads, sizeof(dim3));
    hipMalloc((void**) &D_blocks, sizeof(dim3));


    hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);
    
    //send finish
    hipEventRecord(stop_m);
    
    hipEventSynchronize(stop_m);
    hipEventElapsedTime(&send, start_m, stop_m);

    long* A = D_data;
    long* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    //
    // Slice up the list and give pieces of it to each thread, letting the pieces grow
    // bigger and bigger until the whole list is sorted
    //
    for (int width = 2; width < (size << 1); width <<= 1) {
        long slices = size / ((nThreads) * width) + 1;

        // Actually call the kernel
        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);


        // Switch the input / output arrays instead of copying them around
        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    //
    // Get the list back from the GPU
    //
    //recive start
    hipEventRecord(start_m);
    
    hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost);
    //recieve end
    hipEventRecord(stop_m);
    
    hipEventSynchronize(stop_m);
    
    hipEventElapsedTime(&recv, start_m, stop_m);
    // Free the GPU memory
    hipFree(A);
    hipFree(B);
    
    return send + recv;
}

// GPU helper function
// calculate the id of the current thread
__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

//
// Perform a full mergesort on our section of the data.
//
__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    long start = width*idx*slices, 
         middle, 
         end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

//
// Finally, sort something
// gets called by gpu_mergesort() for each slice
//
__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}

// read data into a minimal linked list
typedef struct {
    int v;
    void* next;
} LinkNode;
